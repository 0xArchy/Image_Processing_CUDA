#include "hip/hip_runtime.h"
/*
 * img_processing.cu -- Image Processing with Kernel filter in CUDA
 * Copyright (c) 2022
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 2.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
 */

#include <stdio.h>
#include <stdlib.h>
#include "functions.h"
#include "hip/hip_runtime.h"
#include "img_processing.h"
#include ""

/*GPU kernel functions*/

__global__ void kernel_filter_rgb(uint8_t *src, uint8_t *dst, int width, int height) {
	int i, j, k, l;
	/* kernel filter */
	int h[3][3] = {{1, 2, 1}, {2, 4, 2}, {1, 2, 1}};

	size_t x = blockIdx.x*blockDim.x + threadIdx.x;
	size_t y = blockIdx.y*blockDim.y + threadIdx.y;

	/* Aplicamos el filtro */
	if (0 < x && x < height-1 && 0 < y && y < 3*width-3) {
		float redval = 0, greenval = 0, blueval = 0;
		for (i = x-1, k = 0 ; i <= x+1 ; i++, k++) {
			for (j = (y*3)-3, l = 0 ; j <= (y*3)+3 ; j+=3, l++) {
				redval += src[(width*3) * i + j]* h[k][l] /16.0;
				greenval += src[(width*3) * i + j+1] * h[k][l] /16.0;
				blueval += src[(width*3) * i + j+2] * h[k][l] /16.0;
			}
		}
		dst[width*3 * x + (y*3)] = redval;
		dst[width*3 * x + (y*3)+1] = greenval;
		dst[width*3 * x + (y*3)+2] = blueval;
	}
}

extern "C" void gpuFilter(uint8_t *src, int width, int height, int loops, color_t imageType)
{
	uint8_t *d_src, *d_dst, *tmp;
	size_t bytes = height * width*3;

    CUDA_SAFE_CALL( hipMalloc(&d_src, bytes * sizeof(uint8_t)) );
    CUDA_SAFE_CALL( hipMalloc(&d_dst, bytes * sizeof(uint8_t)) );
 
    CUDA_SAFE_CALL( hipMemcpy(d_src, src, bytes, hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemset(d_dst, 0, bytes) );

    int t;
	const int blockSize = 16;
	/*Repetimos el bucle loops veces (que indica el usuario)*/
	for (t = 0 ; t < loops ; t++) {
		
		int gridX = FRACTION_CEILING(height, blockSize);
		int gridY = FRACTION_CEILING(width*3, blockSize);
		dim3 block(blockSize, blockSize);
		dim3 grid(gridX, gridY);
		kernel_filter_rgb<<<grid, block>>>(d_src, d_dst, width, height);

		/* intercambiamos los arrays */
		tmp = d_src;
	        d_src = d_dst;
	        d_dst = tmp;
	}

	CUDA_SAFE_CALL( hipGetLastError() );
	CUDA_SAFE_CALL( hipDeviceSynchronize() );
    
    /* Copiamos el array de vuelta al host */
    if (loops%2 == 0) {
    	CUDA_SAFE_CALL( hipMemcpy(src, d_src, bytes, hipMemcpyDeviceToHost) );
    } else {
   		CUDA_SAFE_CALL( hipMemcpy(src, d_dst, bytes, hipMemcpyDeviceToHost) );
   	}

    CUDA_SAFE_CALL( hipFree(d_src) );
    CUDA_SAFE_CALL( hipFree(d_dst) );
}
